#include "hip/hip_runtime.h"
// ====------ test.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

#include "hipblas.h"
#include "nccl.h"
#include "hipDNN.h"
#include "thrust/complex.h"

__global__ void kernel(int *a) {
    *a = clock64();
    __syncthreads();
    int b = *a;
}

void foo() {
    int *a;
    size_t b, c;
    hipDeviceGetPCIBusId(nullptr, 0, 0);
    hipMemGetInfo(&b, &c);
    hipMalloc(&a, sizeof(int));
    hipFree(a);
}