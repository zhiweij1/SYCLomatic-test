// ===-------- text_experimental_obj_memcpy3d_api.cu ----- *- CUDA -* -----===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===---------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iostream>

#define PRINT_PASS 1

using namespace std;

int passed = 0;
int failed = 0;

void checkResult(string name, bool IsPassed) {
  cout << name;
  if (IsPassed) {
    cout << " ---- passed" << endl;
    passed++;
  } else {
    cout << " ---- failed" << endl;
    failed++;
  }
}

__global__ void kernel(short *output, hipTextureObject_t tex, int w, int h,
                       int d) {
  for (int i = 0; i < d; ++i) {
    for (int j = 0; j < h; ++j) {
      for (int k = 0; k < w; ++k) {
        auto ret = tex3D<short4>(tex, k, j, i);
        output[4 * (w * h * i + w * j + k)] = ret.x;
        output[4 * (w * h * i + w * j + k) + 1] = ret.y;
        output[4 * (w * h * i + w * j + k) + 2] = ret.z;
        output[4 * (w * h * i + w * j + k) + 3] = ret.w;
      }
    }
  }
}

hipTextureObject_t getTex(hipArray_t input) {
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = input;

  hipTextureDesc texDesc = {};

  hipTextureObject_t tex;
  hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

  return tex;
}

int main() {
  bool pass = true;

  const int d = 2;
  const int h = 2;
  const int w = 4;
  short4 input[d * h * w] = {
      {1, 2, 3, 4},     {5, 6, 7, 8},
      {9, 10, 11, 12},  {13, 14, 15, 16}, // 1.1
      {17, 18, 19, 20}, {21, 22, 23, 24},
      {25, 26, 27, 28}, {29, 30, 31, 32}, // 1.2
      {33, 34, 35, 36}, {37, 38, 39, 40},
      {41, 42, 43, 44}, {45, 46, 47, 48}, // 2.1
      {49, 50, 51, 52}, {53, 54, 55, 56},
      {57, 58, 59, 60}, {61, 62, 63, 64}, // 2.2
  };
  auto desc = hipCreateChannelDesc<short4>();

  { // p2p
    const auto src = make_hipPitchedPtr(input, w * sizeof(short4), w, h);
    {
      short4 expect[d * h * w] = {
          {1, 2, 3, 4},     {5, 6, 7, 8},
          {9, 10, 11, 12},  {13, 14, 15, 16}, // 1.1
          {17, 18, 19, 20}, {21, 22, 23, 24},
          {25, 26, 27, 28}, {29, 30, 31, 32}, // 1.2
          {33, 34, 35, 36}, {37, 38, 39, 40},
          {41, 42, 43, 44}, {45, 46, 47, 48}, // 2.1
          {49, 50, 51, 52}, {53, 54, 55, 56},
          {57, 58, 59, 60}, {61, 62, 63, 64}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipMemcpy3DParms p = {0};
      p.srcPtr = src;
      p.dstPtr = make_hipPitchedPtr(output, w * sizeof(short4), w, h);
      p.extent = make_hipExtent(w * sizeof(short4), h, d);
      p.kind = hipMemcpyHostToHost;
      hipMemcpy3D(&p);
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3D:p2p:1", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    {
      short4 expect[d * h * w] = {
          {1, 2, 3, 4}, {5, 6, 7, 8}, {9, 10, 11, 12}, {0, 0, 0, 0}, // 1.1
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    {0, 0, 0, 0}, // 1.2
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    {0, 0, 0, 0}, // 2.1
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    {0, 0, 0, 0}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipMemcpy3DParms p = {0};
      p.srcPtr = src;
      p.dstPtr = make_hipPitchedPtr(output, w * sizeof(short4), w, h);
      p.extent = make_hipExtent((w - 1) * sizeof(short4), h - 1, d - 1);
      p.kind = hipMemcpyHostToHost;
      hipMemcpy3D(&p);
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3D:p2p:2", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    {
      short4 expect[d * h * w] = {
          {53, 54, 55, 56}, {57, 58, 59, 60},
          {61, 62, 63, 64}, {0, 0, 0, 0}, // 1.1
          {0, 0, 0, 0},     {0, 0, 0, 0},
          {0, 0, 0, 0},     {0, 0, 0, 0}, // 1.2
          {0, 0, 0, 0},     {0, 0, 0, 0},
          {0, 0, 0, 0},     {0, 0, 0, 0}, // 2.1
          {0, 0, 0, 0},     {0, 0, 0, 0},
          {0, 0, 0, 0},     {0, 0, 0, 0}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipMemcpy3DParms p = {0};
      p.srcPtr = src;
      p.srcPos = make_hipPos(1 * sizeof(short4), 1, 1);
      p.dstPtr = make_hipPitchedPtr(output, w * sizeof(short4), w, h);
      p.extent = make_hipExtent((w - 1) * sizeof(short4), h - 1, d - 1);
      p.kind = hipMemcpyHostToHost;
      hipMemcpy3DAsync(&p);
      hipDeviceSynchronize();
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3DAsync:p2p:3", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    {
      short4 expect[d * h * w] = {
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    // 1.1
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    // 1.2
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    // 2.1
          {0, 0, 0, 0}, {1, 2, 3, 4}, {5, 6, 7, 8}, {9, 10, 11, 12}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipMemcpy3DParms p = {0};
      p.srcPtr = src;
      p.dstPtr = make_hipPitchedPtr(output, w * sizeof(short4), w, h);
      p.dstPos = make_hipPos(1 * sizeof(short4), 1, 1);
      p.extent = make_hipExtent((w - 1) * sizeof(short4), h - 1, d - 1);
      p.kind = hipMemcpyHostToHost;
      hipMemcpy3DAsync(&p);
      hipDeviceSynchronize();
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3DAsync:p2p:4", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    {
      short4 expect[d * h * w] = {
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},     {0, 0, 0, 0},     // 1.1
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},     {0, 0, 0, 0},     // 1.2
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},     {0, 0, 0, 0},     // 2.1
          {0, 0, 0, 0}, {0, 0, 0, 0}, {57, 58, 59, 60}, {61, 62, 63, 64}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipMemcpy3DParms p = {0};
      p.srcPtr = src;
      p.srcPos = make_hipPos(2 * sizeof(short4), 1, 1);
      p.dstPtr = make_hipPitchedPtr(output, w * sizeof(short4), w, h);
      p.dstPos = make_hipPos(2 * sizeof(short4), 1, 1);
      p.extent = make_hipExtent((w - 2) * sizeof(short4), h - 1, d - 1);
      p.kind = hipMemcpyHostToHost;
      hipMemcpy3D(&p);
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3D:p2p:5", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
  }

  { // p2a
    const auto src = make_hipPitchedPtr(input, w * sizeof(short4), w, h);
    {
      short4 expect[d * h * w] = {
          {1, 2, 3, 4},     {5, 6, 7, 8},
          {9, 10, 11, 12},  {13, 14, 15, 16}, // 1.1
          {17, 18, 19, 20}, {21, 22, 23, 24},
          {25, 26, 27, 28}, {29, 30, 31, 32}, // 1.2
          {33, 34, 35, 36}, {37, 38, 39, 40},
          {41, 42, 43, 44}, {45, 46, 47, 48}, // 2.1
          {49, 50, 51, 52}, {53, 54, 55, 56},
          {57, 58, 59, 60}, {61, 62, 63, 64}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipArray *array;
      hipMalloc3DArray(&array, &desc, {w, h, d});
      hipMemcpy3DParms p = {0};
      p.srcPtr = src;
      p.dstArray = array;
      p.extent = make_hipExtent(w, h, d);
      p.kind = hipMemcpyHostToDevice;
      hipMemcpy3D(&p);
      auto tex = getTex(array);
      kernel<<<1, 1>>>(output, tex, w, h, d);
      hipDeviceSynchronize();
      hipDestroyTextureObject(tex);
      hipFreeArray(array);
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3D:p2a:1", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    {
      short4 expect[d * h * w] = {
          {1, 2, 3, 4}, {5, 6, 7, 8}, {9, 10, 11, 12}, {0, 0, 0, 0}, // 1.1
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    {0, 0, 0, 0}, // 1.2
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    {0, 0, 0, 0}, // 2.1
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    {0, 0, 0, 0}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipArray *array;
      hipMalloc3DArray(&array, &desc, {w, h, d});
      hipMemcpy3DParms p = {0};
      p.srcPtr = src;
      p.dstArray = array;
      p.extent = make_hipExtent(w - 1, h - 1, d - 1);
      p.kind = hipMemcpyHostToDevice;
      hipMemcpy3D(&p);
      auto tex = getTex(array);
      kernel<<<1, 1>>>(output, tex, w, h, d);
      hipDeviceSynchronize();
      hipDestroyTextureObject(tex);
      hipFreeArray(array);
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3D:p2a:2", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    {
      short4 expect[d * h * w] = {
          {53, 54, 55, 56}, {57, 58, 59, 60},
          {61, 62, 63, 64}, {0, 0, 0, 0}, // 1.1
          {0, 0, 0, 0},     {0, 0, 0, 0},
          {0, 0, 0, 0},     {0, 0, 0, 0}, // 1.2
          {0, 0, 0, 0},     {0, 0, 0, 0},
          {0, 0, 0, 0},     {0, 0, 0, 0}, // 2.1
          {0, 0, 0, 0},     {0, 0, 0, 0},
          {0, 0, 0, 0},     {0, 0, 0, 0}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipArray *array;
      hipMalloc3DArray(&array, &desc, {w, h, d});
      hipMemcpy3DParms p = {0};
      p.srcPtr = src;
      p.srcPos = make_hipPos(1, 1, 1);
      p.dstArray = array;
      p.extent = make_hipExtent(w - 1, h - 1, d - 1);
      p.kind = hipMemcpyHostToDevice;
      hipMemcpy3DAsync(&p);
      hipDeviceSynchronize();
      auto tex = getTex(array);
      kernel<<<1, 1>>>(output, tex, w, h, d);
      hipDeviceSynchronize();
      hipDestroyTextureObject(tex);
      hipFreeArray(array);
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3DAsync:p2a:3", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    {
      short4 expect[d * h * w] = {
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    // 1.1
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    // 1.2
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    // 2.1
          {0, 0, 0, 0}, {1, 2, 3, 4}, {5, 6, 7, 8}, {9, 10, 11, 12}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipArray *array;
      hipMalloc3DArray(&array, &desc, {w, h, d});
      hipMemcpy3DParms p = {0};
      p.srcPtr = src;
      p.dstArray = array;
      p.dstPos = make_hipPos(1, 1, 1);
      p.extent = make_hipExtent(w - 1, h - 1, d - 1);
      p.kind = hipMemcpyHostToDevice;
      hipMemcpy3DAsync(&p);
      hipDeviceSynchronize();
      auto tex = getTex(array);
      kernel<<<1, 1>>>(output, tex, w, h, d);
      hipDeviceSynchronize();
      hipDestroyTextureObject(tex);
      hipFreeArray(array);
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3DAsync:p2a:4", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    {
      short4 expect[d * h * w] = {
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},     {0, 0, 0, 0},     // 1.1
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},     {0, 0, 0, 0},     // 1.2
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},     {0, 0, 0, 0},     // 2.1
          {0, 0, 0, 0}, {0, 0, 0, 0}, {57, 58, 59, 60}, {61, 62, 63, 64}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipArray *array;
      hipMalloc3DArray(&array, &desc, {w, h, d});
      hipMemcpy3DParms p = {0};
      p.srcPtr = src;
      p.srcPos = make_hipPos(2, 1, 1);
      p.dstArray = array;
      p.dstPos = make_hipPos(2, 1, 1);
      p.extent = make_hipExtent(w - 2, h - 1, d - 1);
      p.kind = hipMemcpyHostToDevice;
      hipMemcpy3D(&p);
      auto tex = getTex(array);
      kernel<<<1, 1>>>(output, tex, w, h, d);
      hipDeviceSynchronize();
      hipDestroyTextureObject(tex);
      hipFreeArray(array);
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3D:p2a:5", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
  }

  { // a2p
    hipArray *src;
    hipMalloc3DArray(&src, &desc, {w, h, d});
    hipMemcpy3DParms p = {0};
    p.srcPtr = make_hipPitchedPtr(input, w * sizeof(short4), w, h);
    p.dstArray = src;
    p.extent = make_hipExtent(w, h, d);
    p.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&p);
    {
      short4 expect[d * h * w] = {
          {1, 2, 3, 4},     {5, 6, 7, 8},
          {9, 10, 11, 12},  {13, 14, 15, 16}, // 1.1
          {17, 18, 19, 20}, {21, 22, 23, 24},
          {25, 26, 27, 28}, {29, 30, 31, 32}, // 1.2
          {33, 34, 35, 36}, {37, 38, 39, 40},
          {41, 42, 43, 44}, {45, 46, 47, 48}, // 2.1
          {49, 50, 51, 52}, {53, 54, 55, 56},
          {57, 58, 59, 60}, {61, 62, 63, 64}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipMemcpy3DParms p = {0};
      p.srcArray = src;
      p.dstPtr = make_hipPitchedPtr(output, w * sizeof(short4), w, h);
      p.extent = make_hipExtent(w, h, d);
      p.kind = hipMemcpyDeviceToHost;
      hipMemcpy3D(&p);
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3D:a2p:1", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    {
      short4 expect[d * h * w] = {
          {1, 2, 3, 4}, {5, 6, 7, 8}, {9, 10, 11, 12}, {0, 0, 0, 0}, // 1.1
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    {0, 0, 0, 0}, // 1.2
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    {0, 0, 0, 0}, // 2.1
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    {0, 0, 0, 0}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipMemcpy3DParms p = {0};
      p.srcArray = src;
      p.dstPtr = make_hipPitchedPtr(output, w * sizeof(short4), w, h);
      p.extent = make_hipExtent(w - 1, h - 1, d - 1);
      p.kind = hipMemcpyDeviceToHost;
      hipMemcpy3D(&p);
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3D:a2p:2", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    {
      short4 expect[d * h * w] = {
          {53, 54, 55, 56}, {57, 58, 59, 60},
          {61, 62, 63, 64}, {0, 0, 0, 0}, // 1.1
          {0, 0, 0, 0},     {0, 0, 0, 0},
          {0, 0, 0, 0},     {0, 0, 0, 0}, // 1.2
          {0, 0, 0, 0},     {0, 0, 0, 0},
          {0, 0, 0, 0},     {0, 0, 0, 0}, // 2.1
          {0, 0, 0, 0},     {0, 0, 0, 0},
          {0, 0, 0, 0},     {0, 0, 0, 0}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipMemcpy3DParms p = {0};
      p.srcArray = src;
      p.srcPos = make_hipPos(1, 1, 1);
      p.dstPtr = make_hipPitchedPtr(output, w * sizeof(short4), w, h);
      p.extent = make_hipExtent(w - 1, h - 1, d - 1);
      p.kind = hipMemcpyDeviceToHost;
      hipMemcpy3DAsync(&p);
      hipDeviceSynchronize();
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3DAsync:a2p:3", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    {
      short4 expect[d * h * w] = {
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    // 1.1
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    // 1.2
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},    // 2.1
          {0, 0, 0, 0}, {1, 2, 3, 4}, {5, 6, 7, 8}, {9, 10, 11, 12}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipMemcpy3DParms p = {0};
      p.srcArray = src;
      p.dstPtr = make_hipPitchedPtr(output, w * sizeof(short4), w, h);
      p.dstPos = make_hipPos(1, 1, 1);
      p.extent = make_hipExtent(w - 1, h - 1, d - 1);
      p.kind = hipMemcpyDeviceToHost;
      hipMemcpy3DAsync(&p);
      hipDeviceSynchronize();
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3DAsync:a2p:4", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    {
      short4 expect[d * h * w] = {
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},     {0, 0, 0, 0},     // 1.1
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},     {0, 0, 0, 0},     // 1.2
          {0, 0, 0, 0}, {0, 0, 0, 0}, {0, 0, 0, 0},     {0, 0, 0, 0},     // 2.1
          {0, 0, 0, 0}, {0, 0, 0, 0}, {57, 58, 59, 60}, {61, 62, 63, 64}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipMemcpy3DParms p = {0};
      p.srcArray = src;
      p.srcPos = make_hipPos(2, 1, 1);
      p.dstPtr = make_hipPitchedPtr(output, w * sizeof(short4), w, h);
      p.dstPos = make_hipPos(2, 1, 1);
      p.extent = make_hipExtent(w - 2, h - 1, d - 1);
      p.kind = hipMemcpyDeviceToHost;
      hipMemcpy3D(&p);
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3D:a2p:5", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    hipFreeArray(src);
  }

  { // a2a
    hipArray *src;
    hipMalloc3DArray(&src, &desc, {w, h, d});
    hipMemcpy3DParms p = {0};
    p.srcPtr = make_hipPitchedPtr(input, w * sizeof(short4), w, h);
    p.dstArray = src;
    p.extent = make_hipExtent(w, h, d);
    p.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&p);
    {
      short4 expect[d * h * w] = {
          {1, 2, 3, 4},     {5, 6, 7, 8},
          {9, 10, 11, 12},  {13, 14, 15, 16}, // 1.1
          {17, 18, 19, 20}, {21, 22, 23, 24},
          {25, 26, 27, 28}, {29, 30, 31, 32}, // 1.2
          {33, 34, 35, 36}, {37, 38, 39, 40},
          {41, 42, 43, 44}, {45, 46, 47, 48}, // 2.1
          {49, 50, 51, 52}, {53, 54, 55, 56},
          {57, 58, 59, 60}, {61, 62, 63, 64}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipArray *array;
      hipMalloc3DArray(&array, &desc, {w, h, d});
      hipMemcpy3DParms p = {0};
      p.srcArray = src;
      p.dstArray = array;
      p.extent = make_hipExtent(w, h, d);
      p.kind = hipMemcpyDeviceToDevice;
      hipMemcpy3D(&p);
      auto tex = getTex(array);
      kernel<<<1, 1>>>(output, tex, w, h, d);
      hipDeviceSynchronize();
      hipDestroyTextureObject(tex);
      hipFreeArray(array);
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3D:a2a:1", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    {
      short4 expect[d * h * w] = {
          {1, 2, 3, 4},     {5, 6, 7, 8},
          {9, 10, 11, 12},  {13, 14, 15, 16}, // 1.1
          {17, 18, 19, 20}, {21, 22, 23, 24},
          {25, 26, 27, 28}, {29, 30, 31, 32}, // 1.2
          {33, 34, 35, 36}, {37, 38, 39, 40},
          {41, 42, 43, 44}, {45, 46, 47, 48}, // 2.1
          {49, 50, 51, 52}, {53, 54, 55, 56},
          {57, 58, 59, 60}, {61, 62, 63, 64}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipArray *array;
      hipMalloc3DArray(&array, &desc, {w, h, d});
      hipMemcpy3DParms p = {0};
      p.srcArray = src;
      p.dstArray = array;
      p.extent = make_hipExtent(w - 1, h - 1, d - 1);
      p.kind = hipMemcpyDeviceToDevice;
      hipMemcpy3D(&p);
      auto tex = getTex(array);
      kernel<<<1, 1>>>(output, tex, w, h, d);
      hipDeviceSynchronize();
      hipDestroyTextureObject(tex);
      hipFreeArray(array);
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3D:a2a:2", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    {
      short4 expect[d * h * w] = {
          {53, 54, 55, 56}, {57, 58, 59, 60},
          {61, 62, 63, 64}, {13, 14, 15, 16}, // 1.1
          {17, 18, 19, 20}, {21, 22, 23, 24},
          {25, 26, 27, 28}, {29, 30, 31, 32}, // 1.2
          {33, 34, 35, 36}, {37, 38, 39, 40},
          {41, 42, 43, 44}, {45, 46, 47, 48}, // 2.1
          {49, 50, 51, 52}, {53, 54, 55, 56},
          {57, 58, 59, 60}, {61, 62, 63, 64}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipArray *array;
      hipMalloc3DArray(&array, &desc, {w, h, d});
      hipMemcpy3DParms p = {0};
      p.srcArray = src;
      p.srcPos = make_hipPos(1, 1, 1);
      p.dstArray = array;
      p.extent = make_hipExtent(w - 1, h - 1, d - 1);
      p.kind = hipMemcpyDeviceToDevice;
      hipMemcpy3DAsync(&p);
      hipDeviceSynchronize();
      auto tex = getTex(array);
      kernel<<<1, 1>>>(output, tex, w, h, d);
      hipDeviceSynchronize();
      hipDestroyTextureObject(tex);
      hipFreeArray(array);
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3DAsync:a2a:3", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    {
      short4 expect[d * h * w] = {
          {53, 54, 55, 56}, {57, 58, 59, 60},
          {61, 62, 63, 64}, {13, 14, 15, 16}, // 1.1
          {17, 18, 19, 20}, {21, 22, 23, 24},
          {25, 26, 27, 28}, {29, 30, 31, 32}, // 1.2
          {33, 34, 35, 36}, {37, 38, 39, 40},
          {41, 42, 43, 44}, {45, 46, 47, 48}, // 2.1
          {49, 50, 51, 52}, {1, 2, 3, 4},
          {5, 6, 7, 8},     {9, 10, 11, 12}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipArray *array;
      hipMalloc3DArray(&array, &desc, {w, h, d});
      hipMemcpy3DParms p = {0};
      p.srcArray = src;
      p.dstArray = array;
      p.dstPos = make_hipPos(1, 1, 1);
      p.extent = make_hipExtent(w - 1, h - 1, d - 1);
      p.kind = hipMemcpyDeviceToDevice;
      hipMemcpy3DAsync(&p);
      hipDeviceSynchronize();
      auto tex = getTex(array);
      kernel<<<1, 1>>>(output, tex, w, h, d);
      hipDeviceSynchronize();
      hipDestroyTextureObject(tex);
      hipFreeArray(array);
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3DAsync:a2a:4", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    {
      short4 expect[d * h * w] = {
          {53, 54, 55, 56}, {57, 58, 59, 60},
          {61, 62, 63, 64}, {13, 14, 15, 16}, // 1.1
          {17, 18, 19, 20}, {21, 22, 23, 24},
          {25, 26, 27, 28}, {29, 30, 31, 32}, // 1.2
          {33, 34, 35, 36}, {37, 38, 39, 40},
          {41, 42, 43, 44}, {45, 46, 47, 48}, // 2.1
          {49, 50, 51, 52}, {1, 2, 3, 4},
          {57, 58, 59, 60}, {61, 62, 63, 64}, // 2.2
      };
      short *output;
      hipMallocManaged(&output, sizeof(expect));
      hipArray *array;
      hipMalloc3DArray(&array, &desc, {w, h, d});
      hipMemcpy3DParms p = {0};
      p.srcArray = src;
      p.srcPos = make_hipPos(2, 1, 1);
      p.dstArray = array;
      p.dstPos = make_hipPos(2, 1, 1);
      p.extent = make_hipExtent(w - 2, h - 1, d - 1);
      p.kind = hipMemcpyDeviceToDevice;
      hipMemcpy3D(&p);
      auto tex = getTex(array);
      kernel<<<1, 1>>>(output, tex, w, h, d);
      hipDeviceSynchronize();
      hipDestroyTextureObject(tex);
      hipFreeArray(array);
      for (int i = 0; i < w * h * d; ++i) {
        if (output[4 * i] != expect[i].x || output[4 * i + 1] != expect[i].y ||
            output[4 * i + 2] != expect[i].z ||
            output[4 * i + 3] != expect[i].w) {
          pass = false;
          break;
        }
      }
      checkResult("cudaMemcpy3D:a2a:5", pass);
      if (PRINT_PASS || !pass)
        for (int i = 0; i < d; ++i) {
          for (int j = 0; j < h; ++j) {
            for (int k = 0; k < w; ++k)
              cout << "{" << output[4 * (w * h * i + j * w + k)] << ", "
                   << output[4 * (w * h * i + j * w + k) + 1] << ", "
                   << output[4 * (w * h * i + j * w + k) + 2] << ", "
                   << output[4 * (w * h * i + j * w + k) + 3] << "}, ";
            cout << endl;
          }
          cout << endl;
        }
      pass = true;
    }
    hipFreeArray(src);
  }

  cout << "passed " << passed << "/" << passed + failed << " cases!" << endl;
  if (failed) {
    cout << "failed!" << endl;
  }
  return failed;
}
