// ===------------ matmul.cu ----------------------------- *- CUDA -* ----=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //


#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <cstdint>
#include <stdexcept>

const constexpr int COL_TURING = 0;
const constexpr int COL_AMPERE = 1;

// The original source of below two functions was under the license below:
// Copyright (c) Facebook, Inc. and its affiliates.
//
// This source code is licensed under the MIT license found in the
// LICENSE file in the root directory of this source tree.
//
// Repo: https://github.com/TimDettmers/bitsandbytes.git
inline int checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS API failed with status %d\n", status);
        //throw std::logic_error("cuBLAS API failed");
        return 1;
    }
    return 0;
}

template <int FORMATB, int DTYPE_OUT, int SCALE_ROWS> int igemmlt(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc)
{
    int has_error = 0;
    hipblasLtMatmulDesc_t matmulDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    hipblasOperation_t opT = HIPBLAS_OP_T;
    hipblasLtPointerMode_t alphaVec = CUBLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_ZERO;
    hipblasLtOrder_t col32 = CUBLASLT_ORDER_COL32;
    hipblasLtOrder_t col_turing = CUBLASLT_ORDER_COL4_4R2_8C;
    hipblasLtOrder_t col_ampere = CUBLASLT_ORDER_COL32_2R_4R4;

    has_error |= checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8I, m, k, lda));
    has_error |= checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8I, n, k, ldb));

    has_error |= checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &col32, sizeof(col32)));
    if(FORMATB == COL_TURING)
      has_error |= checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &col_turing, sizeof(col_turing)));
    else
      has_error |= checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &col_ampere, sizeof(col_ampere)));

    if(DTYPE_OUT == 32)
    {
      has_error |= checkCublasStatus(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32I, HIP_R_32I));
      has_error |= checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opT, sizeof(opT)));
      has_error |= checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32I, m, n, ldc));
      has_error |= checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &col32, sizeof(col32)));
      int alpha = 1, beta = 0;
      has_error |= checkCublasStatus(hipblasLtMatmul(ltHandle, matmulDesc,&alpha, A, Adesc, B, Bdesc, &beta, (int32_t*)C, Cdesc, (int32_t*)C, Cdesc, NULL, NULL, 0, 0));
    }
    else
    {
      has_error |= checkCublasStatus(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32I, HIP_R_32F));
      has_error |= checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opT, sizeof(opT)));
      has_error |= checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_8I, m, n, ldc));
      has_error |= checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &col32, sizeof(col32)));
      if(!SCALE_ROWS)
      {
        float alpha = 1.0f, beta = 0.0f;
        has_error |= checkCublasStatus(hipblasLtMatmul(ltHandle, matmulDesc,&alpha, A, Adesc, B, Bdesc, &beta, (int8_t*)C, Cdesc, (int8_t*)C, Cdesc, NULL, NULL, 0, 0));
      }
      else
      {
        has_error |= checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_POINTER_MODE, &alphaVec, sizeof(alphaVec)));
        has_error |= checkCublasStatus(hipblasLtMatmul(ltHandle, matmulDesc, row_scale, A, Adesc, B, Bdesc, NULL, (int8_t*)C, Cdesc, (int8_t*)C, Cdesc, NULL, NULL, 0, 0));
      }
    }

    hipStreamSynchronize(0);

    if (Cdesc) has_error |= checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) has_error |= checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) has_error |= checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (matmulDesc) has_error |= checkCublasStatus(hipblasLtMatmulDescDestroy(matmulDesc));
    if(has_error == 1)
      printf("error detected");

    return has_error;
}

void transform(hipblasLtHandle_t ltHandle, const void *in, int ld_in,
               hipblasLtMatrixLayout_t layout_in, void *out, int ld_out,
               hipblasLtMatrixLayout_t layout_out) {
  hipblasLtMatrixTransformDesc_t transform_desc = NULL;
  hipblasLtMatrixTransformDescCreate(&transform_desc, HIP_R_32F);
  float alpha = 1.0f, beta = 0.0f;
  hipblasLtMatrixTransform(ltHandle, transform_desc, &alpha, in, layout_in,
                          &beta, NULL, NULL, out, layout_out, 0);
  hipblasLtMatrixTransformDescDestroy(transform_desc);
}

// igemmlt<COL_TURING, 8, 0>
bool test2() {
  hipblasLtHandle_t ltHandle;
  hipblasLtCreate(&ltHandle);
  const constexpr int m = 4;
  const constexpr int n = 2;
  const constexpr int k = 3;
  int lda = m;
  int ldb = n;
  int ldc = m;
  void *Adev;
  void *Bdev;
  void *Cdev;
  hipMalloc(&Adev, m * k * sizeof(int8_t));
  hipMalloc(&Bdev, n * k * sizeof(int8_t));
  hipMalloc(&Cdev, m * n * sizeof(int8_t));

  int8_t Ahost[m * k] = {6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17};
  int8_t Bhost[n * k] = {5, 4, -3, -2, 1, 0};

  hipMemcpy(Adev, Ahost, m * k * sizeof(int8_t), hipMemcpyHostToDevice);
  hipMemcpy(Bdev, Bhost, n * k * sizeof(int8_t), hipMemcpyHostToDevice);

  hipblasLtMatrixLayout_t Adesc_col_major = NULL, Bdesc_col_major = NULL,
                         Cdesc_col_major = NULL;
  hipblasLtMatrixLayoutCreate(&Adesc_col_major, HIP_R_8I, m, k, lda);
  hipblasLtMatrixLayoutCreate(&Bdesc_col_major, HIP_R_8I, n, k, ldb);
  hipblasLtMatrixLayoutCreate(&Cdesc_col_major, HIP_R_8I, m, n, ldc);

  // Convert A and B
  hipblasLtMatrixLayout_t Adesc_col32 = NULL, Bdesc_col4_4r2_8c = NULL,
                         Cdesc_col32 = NULL;
  int8_t *A_col32, *B_col4_4r2_8c;
  int8_t *C_col32;
  hipMalloc(&A_col32, m * 32 * sizeof(std::int8_t));
  hipMalloc(&B_col4_4r2_8c, ((n + 8 - 1) / 8) * 8 * 32 * sizeof(std::int8_t));
  hipMalloc(&C_col32, m * 32 * sizeof(std::int8_t));
  hipblasLtMatrixLayoutCreate(&Adesc_col32, HIP_R_8I, m, k, m * 32);
  hipblasLtMatrixLayoutCreate(&Bdesc_col4_4r2_8c, HIP_R_8I, k, n,
                             ((n + 8 - 1) / 8) * 8 * 32);
  hipblasLtMatrixLayoutCreate(&Cdesc_col32, HIP_R_8I, m, n, m * 32);
  hipblasLtOrder_t col32 = CUBLASLT_ORDER_COL32;
  hipblasLtOrder_t col4_4r2_8c = CUBLASLT_ORDER_COL4_4R2_8C;
  hipblasLtMatrixLayoutSetAttribute(Adesc_col32, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                   &col32, sizeof(col32));
  hipblasLtMatrixLayoutSetAttribute(Bdesc_col4_4r2_8c,
                                   HIPBLASLT_MATRIX_LAYOUT_ORDER, &col4_4r2_8c,
                                   sizeof(col4_4r2_8c));
  hipblasLtMatrixLayoutSetAttribute(Cdesc_col32, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                   &col32, sizeof(col32));

  transform(ltHandle, Adev, lda, Adesc_col_major, A_col32, m * 32, Adesc_col32);
  transform(ltHandle, Bdev, ldb, Bdesc_col_major, B_col4_4r2_8c, 8 * 32,
            Bdesc_col4_4r2_8c);

  // Matmul
  igemmlt<COL_TURING, 8, 0>(ltHandle, m, n, k, A_col32, B_col4_4r2_8c, C_col32,
                            nullptr, m * 32, ((n + 8 - 1) / 8) * 8 * 32,
                            m * 32);

  // Convert C
  transform(ltHandle, C_col32, m * 32, Cdesc_col32, Cdev, ldc, Cdesc_col_major);
  hipStreamSynchronize(0);

  // Check result
  int8_t Chost[m * n];
  hipMemcpy(Chost, Cdev, m * n * sizeof(int8_t), hipMemcpyDeviceToHost);

  bool error = false;
  int8_t C_ref[m * n] = {14, 17, 20, 23, 4, 6, 8, 10};
  for (int i = 0; i < m * n; i++) {
    if (Chost[i] != C_ref[i]) {
      error = true;
      break;
    }
  }
  printf("c:\n");
  for (int i = 0; i < m * n; i++)
    printf("%d, ", Chost[i]);
  printf("\n");

  if (error) {
    printf("error\n");
  } else {
    printf("success\n");
  }

  hipblasLtDestroy(ltHandle);
  hipblasLtMatrixLayoutDestroy(Adesc_col32);
  hipblasLtMatrixLayoutDestroy(Bdesc_col4_4r2_8c);
  hipblasLtMatrixLayoutDestroy(Cdesc_col32);
  hipblasLtMatrixLayoutDestroy(Adesc_col_major);
  hipblasLtMatrixLayoutDestroy(Bdesc_col_major);
  hipblasLtMatrixLayoutDestroy(Cdesc_col_major);
  hipFree(Adev);
  hipFree(Bdev);
  hipFree(Cdev);
  hipFree(A_col32);
  hipFree(B_col4_4r2_8c);
  hipFree(C_col32);

  return !error;
}

// igemmlt<COL_TURING, 8, 1>
bool test3() {
  hipblasLtHandle_t ltHandle;
  hipblasLtCreate(&ltHandle);
  const constexpr int m = 4;
  const constexpr int n = 2;
  const constexpr int k = 3;
  int lda = m;
  int ldb = n;
  int ldc = m;
  void *Adev;
  void *Bdev;
  void *Cdev;
  hipMalloc(&Adev, m * k * sizeof(int8_t));
  hipMalloc(&Bdev, n * k * sizeof(int8_t));
  hipMalloc(&Cdev, m * n * sizeof(int8_t));

  int8_t Ahost[m * k] = {6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17};
  int8_t Bhost[n * k] = {5, 4, -3, -2, 1, 0};

  hipMemcpy(Adev, Ahost, m * k * sizeof(int8_t), hipMemcpyHostToDevice);
  hipMemcpy(Bdev, Bhost, n * k * sizeof(int8_t), hipMemcpyHostToDevice);

  hipblasLtMatrixLayout_t Adesc_col_major = NULL, Bdesc_col_major = NULL,
                         Cdesc_col_major = NULL;
  hipblasLtMatrixLayoutCreate(&Adesc_col_major, HIP_R_8I, m, k, lda);
  hipblasLtMatrixLayoutCreate(&Bdesc_col_major, HIP_R_8I, n, k, ldb);
  hipblasLtMatrixLayoutCreate(&Cdesc_col_major, HIP_R_8I, m, n, ldc);

  // Convert A and B
  hipblasLtMatrixLayout_t Adesc_col32 = NULL, Bdesc_col4_4r2_8c = NULL,
                         Cdesc_col32 = NULL;
  int8_t *A_col32, *B_col4_4r2_8c;
  int8_t *C_col32;
  hipMalloc(&A_col32, m * 32 * sizeof(std::int8_t));
  hipMalloc(&B_col4_4r2_8c, ((n + 8 - 1) / 8) * 8 * 32 * sizeof(std::int8_t));
  hipMalloc(&C_col32, m * 32 * sizeof(std::int8_t));
  hipblasLtMatrixLayoutCreate(&Adesc_col32, HIP_R_8I, m, k, m * 32);
  hipblasLtMatrixLayoutCreate(&Bdesc_col4_4r2_8c, HIP_R_8I, k, n,
                             ((n + 8 - 1) / 8) * 8 * 32);
  hipblasLtMatrixLayoutCreate(&Cdesc_col32, HIP_R_8I, m, n, m * 32);
  hipblasLtOrder_t col32 = CUBLASLT_ORDER_COL32;
  hipblasLtOrder_t col4_4r2_8c = CUBLASLT_ORDER_COL4_4R2_8C;
  hipblasLtMatrixLayoutSetAttribute(Adesc_col32, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                   &col32, sizeof(col32));
  hipblasLtMatrixLayoutSetAttribute(Bdesc_col4_4r2_8c,
                                   HIPBLASLT_MATRIX_LAYOUT_ORDER, &col4_4r2_8c,
                                   sizeof(col4_4r2_8c));
  hipblasLtMatrixLayoutSetAttribute(Cdesc_col32, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                   &col32, sizeof(col32));

  transform(ltHandle, Adev, lda, Adesc_col_major, A_col32, m * 32, Adesc_col32);
  transform(ltHandle, Bdev, ldb, Bdesc_col_major, B_col4_4r2_8c, 8 * 32,
            Bdesc_col4_4r2_8c);

  float *alpha;
  hipMallocManaged(&alpha, 4 * sizeof(float));
  alpha[0] = 0;
  alpha[1] = 1;
  alpha[2] = 2;
  alpha[3] = 3;

  // Matmul
  igemmlt<COL_TURING, 8, 1>(ltHandle, m, n, k, A_col32, B_col4_4r2_8c, C_col32,
                            alpha, m * 32, ((n + 8 - 1) / 8) * 8 * 32, m * 32);

  // Convert C
  transform(ltHandle, C_col32, m * 32, Cdesc_col32, Cdev, ldc, Cdesc_col_major);
  hipStreamSynchronize(0);

  // Check result
  int8_t Chost[m * n];
  hipMemcpy(Chost, Cdev, m * n * sizeof(int8_t), hipMemcpyDeviceToHost);

  bool error = false;
  int8_t C_ref[m * n] = {0, 17, 40, 69, 0, 6, 16, 30};
  for (int i = 0; i < m * n; i++) {
    if (Chost[i] != C_ref[i]) {
      error = true;
      break;
    }
  }
  printf("c:\n");
  for (int i = 0; i < m * n; i++)
    printf("%d, ", Chost[i]);
  printf("\n");

  if (error) {
    printf("error\n");
  } else {
    printf("success\n");
  }

  hipblasLtDestroy(ltHandle);
  hipblasLtMatrixLayoutDestroy(Adesc_col32);
  hipblasLtMatrixLayoutDestroy(Bdesc_col4_4r2_8c);
  hipblasLtMatrixLayoutDestroy(Cdesc_col32);
  hipblasLtMatrixLayoutDestroy(Adesc_col_major);
  hipblasLtMatrixLayoutDestroy(Bdesc_col_major);
  hipblasLtMatrixLayoutDestroy(Cdesc_col_major);
  hipFree(Adev);
  hipFree(Bdev);
  hipFree(Cdev);
  hipFree(alpha);
  hipFree(A_col32);
  hipFree(B_col4_4r2_8c);
  hipFree(C_col32);

  return !error;
}

// igemmlt<COL_AMPERE, 8, 0>
bool test5() {
  hipblasLtHandle_t ltHandle;
  hipblasLtCreate(&ltHandle);
  const constexpr int m = 4;
  const constexpr int n = 2;
  const constexpr int k = 3;
  int lda = m;
  int ldb = n;
  int ldc = m;
  void *Adev;
  void *Bdev;
  void *Cdev;
  hipMalloc(&Adev, m * k * sizeof(int8_t));
  hipMalloc(&Bdev, n * k * sizeof(int8_t));
  hipMalloc(&Cdev, m * n * sizeof(int8_t));

  int8_t Ahost[m * k] = {6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17};
  int8_t Bhost[n * k] = {5, 4, -3, -2, 1, 0};

  hipMemcpy(Adev, Ahost, m * k * sizeof(int8_t), hipMemcpyHostToDevice);
  hipMemcpy(Bdev, Bhost, n * k * sizeof(int8_t), hipMemcpyHostToDevice);

  hipblasLtMatrixLayout_t Adesc_col_major = NULL, Bdesc_col_major = NULL,
                         Cdesc_col_major = NULL;
  hipblasLtMatrixLayoutCreate(&Adesc_col_major, HIP_R_8I, m, k, lda);
  hipblasLtMatrixLayoutCreate(&Bdesc_col_major, HIP_R_8I, n, k, ldb);
  hipblasLtMatrixLayoutCreate(&Cdesc_col_major, HIP_R_8I, m, n, ldc);

  // Convert A and B
  hipblasLtMatrixLayout_t Adesc_col32 = NULL, Bdesc_col32_2r_4r4 = NULL,
                         Cdesc_col32 = NULL;
  int8_t *A_col32, *B_col32_2r_4r4;
  int8_t *C_col32;
  hipMalloc(&A_col32, m * 32 * sizeof(std::int8_t));
  hipMalloc(&B_col32_2r_4r4,
             ((n + 32 - 1) / 32) * 32 * 32 * sizeof(std::int8_t));
  hipMalloc(&C_col32, m * 32 * sizeof(std::int8_t));
  hipblasLtMatrixLayoutCreate(&Adesc_col32, HIP_R_8I, m, k, m * 32);
  hipblasLtMatrixLayoutCreate(&Bdesc_col32_2r_4r4, HIP_R_8I, k, n,
                             ((n + 32 - 1) / 32) * 32 * 32);
  hipblasLtMatrixLayoutCreate(&Cdesc_col32, HIP_R_8I, m, n, m * 32);
  hipblasLtOrder_t col32 = CUBLASLT_ORDER_COL32;
  hipblasLtOrder_t col32_2r_4r4 = CUBLASLT_ORDER_COL32_2R_4R4;
  hipblasLtMatrixLayoutSetAttribute(Adesc_col32, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                   &col32, sizeof(col32));
  hipblasLtMatrixLayoutSetAttribute(Bdesc_col32_2r_4r4,
                                   HIPBLASLT_MATRIX_LAYOUT_ORDER, &col32_2r_4r4,
                                   sizeof(col32_2r_4r4));
  hipblasLtMatrixLayoutSetAttribute(Cdesc_col32, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                   &col32, sizeof(col32));

  transform(ltHandle, Adev, lda, Adesc_col_major, A_col32, m * 32, Adesc_col32);
  transform(ltHandle, Bdev, ldb, Bdesc_col_major, B_col32_2r_4r4, 8 * 32,
            Bdesc_col32_2r_4r4);

  // Matmul
  igemmlt<COL_AMPERE, 8, 0>(ltHandle, m, n, k, A_col32, B_col32_2r_4r4, C_col32,
                            nullptr, m * 32, ((n + 8 - 1) / 8) * 8 * 32,
                            m * 32);

  // Convert C
  transform(ltHandle, C_col32, m * 32, Cdesc_col32, Cdev, ldc, Cdesc_col_major);
  hipStreamSynchronize(0);

  // Check result
  int8_t Chost[m * n];
  hipMemcpy(Chost, Cdev, m * n * sizeof(int8_t), hipMemcpyDeviceToHost);

  bool error = false;
  int8_t C_ref[m * n] = {14, 17, 20, 23, 4, 6, 8, 10};
  for (int i = 0; i < m * n; i++) {
    if (Chost[i] != C_ref[i]) {
      error = true;
      break;
    }
  }
  printf("c:\n");
  for (int i = 0; i < m * n; i++)
    printf("%d, ", Chost[i]);
  printf("\n");

  if (error) {
    printf("error\n");
  } else {
    printf("success\n");
  }

  hipblasLtDestroy(ltHandle);
  hipblasLtMatrixLayoutDestroy(Adesc_col32);
  hipblasLtMatrixLayoutDestroy(Bdesc_col32_2r_4r4);
  hipblasLtMatrixLayoutDestroy(Cdesc_col32);
  hipblasLtMatrixLayoutDestroy(Adesc_col_major);
  hipblasLtMatrixLayoutDestroy(Bdesc_col_major);
  hipblasLtMatrixLayoutDestroy(Cdesc_col_major);
  hipFree(Adev);
  hipFree(Bdev);
  hipFree(Cdev);
  hipFree(A_col32);
  hipFree(B_col32_2r_4r4);
  hipFree(C_col32);

  return !error;
}

// igemmlt<COL_AMPERE, 8, 1>
bool test6() {
  hipblasLtHandle_t ltHandle;
  hipblasLtCreate(&ltHandle);
  const constexpr int m = 4;
  const constexpr int n = 2;
  const constexpr int k = 3;
  int lda = m;
  int ldb = n;
  int ldc = m;
  void *Adev;
  void *Bdev;
  void *Cdev;
  hipMalloc(&Adev, m * k * sizeof(int8_t));
  hipMalloc(&Bdev, n * k * sizeof(int8_t));
  hipMalloc(&Cdev, m * n * sizeof(int8_t));

  int8_t Ahost[m * k] = {6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17};
  int8_t Bhost[n * k] = {5, 4, -3, -2, 1, 0};

  hipMemcpy(Adev, Ahost, m * k * sizeof(int8_t), hipMemcpyHostToDevice);
  hipMemcpy(Bdev, Bhost, n * k * sizeof(int8_t), hipMemcpyHostToDevice);

  hipblasLtMatrixLayout_t Adesc_col_major = NULL, Bdesc_col_major = NULL,
                         Cdesc_col_major = NULL;
  hipblasLtMatrixLayoutCreate(&Adesc_col_major, HIP_R_8I, m, k, lda);
  hipblasLtMatrixLayoutCreate(&Bdesc_col_major, HIP_R_8I, n, k, ldb);
  hipblasLtMatrixLayoutCreate(&Cdesc_col_major, HIP_R_8I, m, n, ldc);

  // Convert A and B
  hipblasLtMatrixLayout_t Adesc_col32 = NULL, Bdesc_col32_2r_4r4 = NULL,
                         Cdesc_col32 = NULL;
  int8_t *A_col32, *B_col32_2r_4r4;
  int8_t *C_col32;
  hipMalloc(&A_col32, m * 32 * sizeof(std::int8_t));
  hipMalloc(&B_col32_2r_4r4,
             ((n + 32 - 1) / 32) * 32 * 32 * sizeof(std::int8_t));
  hipMalloc(&C_col32, m * 32 * sizeof(std::int8_t));
  hipblasLtMatrixLayoutCreate(&Adesc_col32, HIP_R_8I, m, k, m * 32);
  hipblasLtMatrixLayoutCreate(&Bdesc_col32_2r_4r4, HIP_R_8I, k, n,
                             ((n + 32 - 1) / 32) * 32 * 32);
  hipblasLtMatrixLayoutCreate(&Cdesc_col32, HIP_R_8I, m, n, m * 32);
  hipblasLtOrder_t col32 = CUBLASLT_ORDER_COL32;
  hipblasLtOrder_t col32_2r_4r4 = CUBLASLT_ORDER_COL32_2R_4R4;
  hipblasLtMatrixLayoutSetAttribute(Adesc_col32, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                   &col32, sizeof(col32));
  hipblasLtMatrixLayoutSetAttribute(Bdesc_col32_2r_4r4,
                                   HIPBLASLT_MATRIX_LAYOUT_ORDER, &col32_2r_4r4,
                                   sizeof(col32_2r_4r4));
  hipblasLtMatrixLayoutSetAttribute(Cdesc_col32, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                   &col32, sizeof(col32));

  transform(ltHandle, Adev, lda, Adesc_col_major, A_col32, m * 32, Adesc_col32);
  transform(ltHandle, Bdev, ldb, Bdesc_col_major, B_col32_2r_4r4, 8 * 32,
            Bdesc_col32_2r_4r4);

  float *alpha;
  hipMallocManaged(&alpha, 4 * sizeof(float));
  alpha[0] = 0;
  alpha[1] = 1;
  alpha[2] = 2;
  alpha[3] = 3;

  // Matmul
  igemmlt<COL_AMPERE, 8, 1>(ltHandle, m, n, k, A_col32, B_col32_2r_4r4, C_col32,
                            alpha, m * 32, ((n + 8 - 1) / 8) * 8 * 32, m * 32);

  // Convert C
  transform(ltHandle, C_col32, m * 32, Cdesc_col32, Cdev, ldc, Cdesc_col_major);
  hipStreamSynchronize(0);

  // Check result
  int8_t Chost[m * n];
  hipMemcpy(Chost, Cdev, m * n * sizeof(int8_t), hipMemcpyDeviceToHost);

  bool error = false;
  int8_t C_ref[m * n] = {0, 17, 40, 69, 0, 6, 16, 30};
  for (int i = 0; i < m * n; i++) {
    if (Chost[i] != C_ref[i]) {
      error = true;
      break;
    }
  }
  printf("c:\n");
  for (int i = 0; i < m * n; i++)
    printf("%d, ", Chost[i]);
  printf("\n");

  if (error) {
    printf("error\n");
  } else {
    printf("success\n");
  }

  hipblasLtDestroy(ltHandle);
  hipblasLtMatrixLayoutDestroy(Adesc_col32);
  hipblasLtMatrixLayoutDestroy(Bdesc_col32_2r_4r4);
  hipblasLtMatrixLayoutDestroy(Cdesc_col32);
  hipblasLtMatrixLayoutDestroy(Adesc_col_major);
  hipblasLtMatrixLayoutDestroy(Bdesc_col_major);
  hipblasLtMatrixLayoutDestroy(Cdesc_col_major);
  hipFree(Adev);
  hipFree(Bdev);
  hipFree(Cdev);
  hipFree(alpha);
  hipFree(A_col32);
  hipFree(B_col32_2r_4r4);
  hipFree(C_col32);

  return !error;
}

void fgemmlt(hipblasLtHandle_t ltHandle, int m, int n, int k,
             const float *A, const float *B, const float *C, float *D,
             float *alpha, float *beta,
             int lda, int ldb, int ldc, int ldd,
             hipblasLtMatrixLayout_t Adesc,
             hipblasLtMatrixLayout_t Bdesc,
             hipblasLtMatrixLayout_t Cdesc,
             hipblasLtMatrixLayout_t Ddesc,
             float *amax_d) {
  hipblasLtMatmulDesc_t matmulDesc = NULL;
  hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);

  float *scale_a;
  float *scale_b;
  float *scale_d;
  hipMallocManaged(&scale_a, sizeof(float));
  hipMallocManaged(&scale_b, sizeof(float));
  hipMallocManaged(&scale_d, sizeof(float));
  scale_a[0] = 3;
  scale_b[0] = 5;
  scale_d[0] = 7;

  hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, &scale_a, sizeof(scale_a));
  hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER, &scale_b, sizeof(scale_b));
  hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER, &scale_d, sizeof(scale_d));
  hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_AMAX_D_POINTER, &amax_d, sizeof(amax_d));

  hipblasLtEpilogue_t ep = HIPBLASLT_EPILOGUE_RELU;
  hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &ep, sizeof(ep));

  hipblasLtMatmul(ltHandle, matmulDesc, alpha, A, Adesc, B, Bdesc, beta, C, Cdesc, D, Ddesc, NULL, NULL, 0, 0);

  hipStreamSynchronize(0);
  hipblasLtMatmulDescDestroy(matmulDesc);
  hipFree(scale_a);
  hipFree(scale_b);
  hipFree(scale_d);
}

// clang-format off
// A (4*3)     B (3*2)
// 6 10 14     5  4
// 7 11 15    -3 -2
// 8 12 16     1  0
// 9 13 17     p  p
//
// alpha * A          * B    + C            = alpha * A*B    + C           = D
// 2*3*5   6  10  14    5  4  -10000 -5000       30   14  4   -10000 -5000  -9580  -4880
//         7  11  15   -3 -2    2000  6000            17  6     2000  6000   2510   6180
//         8  12  16    1  0    3000  7000            20  8     3000  7000   3600   7240
//         9  13  17    p  p    4000  8000            23  10    4000  8000   4690   8300
// scale_d *  D           =  D
//       7 * -9580 -4880    -67060  -34160
//            2510  6180     17570   43260
//            3600  7240     25200   50680
//            4690  8300     32830   58100
// clang-format on

bool test7() {
  hipblasLtHandle_t ltHandle;
  hipblasLtCreate(&ltHandle);
  const constexpr int m = 4;
  const constexpr int n = 2;
  const constexpr int k = 3;
  const constexpr int lda = m;
  const constexpr int ldb = m;
  const constexpr int ldc = m;
  const constexpr int ldd = m;
  void *Adev;
  void *Bdev;
  void *Cdev;
  void *Ddev;
  hipMalloc(&Adev, lda * k * sizeof(float));
  hipMalloc(&Bdev, ldb * n * sizeof(float));
  hipMalloc(&Cdev, ldc * n * sizeof(float));
  hipMalloc(&Ddev, ldd * n * sizeof(float));

  float Ahost[lda * k] = {6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17};
  float Bhost[ldb * n] = {5, -3, 1, 99, 4, -2, 0, 99};
  float Chost[ldc * n] = {-1000, 2000, 3000, 4000, -5000, 6000, 7000, 8000};

  hipMemcpy(Adev, Ahost, lda * k * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Bdev, Bhost, ldb * n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Cdev, Chost, ldc * n * sizeof(float), hipMemcpyHostToDevice);

  hipblasLtMatrixLayout_t Adesc_col_major = NULL,
                         Bdesc_col_major = NULL,
                         Cdesc_col_major = NULL,
                         Ddesc_col_major = NULL;
  hipblasLtMatrixLayoutCreate(&Adesc_col_major, HIP_R_32F, m, k, lda);
  hipblasLtMatrixLayoutCreate(&Bdesc_col_major, HIP_R_32F, k, n, ldb);
  hipblasLtMatrixLayoutCreate(&Cdesc_col_major, HIP_R_32F, m, n, ldc);
  hipblasLtMatrixLayoutCreate(&Ddesc_col_major, HIP_R_32F, m, n, ldd);

  float alpha = 2;
  float beta = 1;

  // Matmul

  float *amax_d;
  hipMallocManaged(&amax_d, sizeof(float));

  fgemmlt(ltHandle, m, n, k, (const float *)Adev, (const float *)Bdev, (const float *)Cdev, (float *)Ddev,
          &alpha, &beta, lda, ldb, ldc, ldd, Adesc_col_major, Bdesc_col_major, Cdesc_col_major, Ddesc_col_major, amax_d);
  hipStreamSynchronize(0);

  // Check result
  float Dhost[ldd * n];
  hipMemcpy(Dhost, Ddev, ldd * n * sizeof(float), hipMemcpyDeviceToHost);

  bool error = false;
  float D_ref[ldd * n] = {0, 17570, 25200, 32830, 0, 43260, 50680, 58100};
  for (int i = 0; i < ldd * n; i++) {
    if (Dhost[i] != D_ref[i]) {
      error = true;
      break;
    }
  }
  if (*amax_d != 8300)
    error = true;

  printf("d:\n");
  for (int i = 0; i < ldd * n; i++)
    printf("%f, ", Dhost[i]);
  printf("\n");
  printf("amax_d:%f\n", *amax_d);

  if (error) {
    printf("error\n");
  } else {
    printf("success\n");
  }

  hipblasLtDestroy(ltHandle);
  hipblasLtMatrixLayoutDestroy(Adesc_col_major);
  hipblasLtMatrixLayoutDestroy(Bdesc_col_major);
  hipblasLtMatrixLayoutDestroy(Cdesc_col_major);
  hipblasLtMatrixLayoutDestroy(Ddesc_col_major);
  hipFree(Adev);
  hipFree(Bdev);
  hipFree(Cdev);
  hipFree(Ddev);
  hipFree(amax_d);

  return !error;
}

bool test_version() {
  size_t v = cublasLtGetVersion();
  printf("version: %lu\n", v);
  printf("test_version pass\n");
  return true;
}

// clang-format off
// A (4*3)    B (2*3)
// 6 10 14    5 -3 1
// 7 11 15    4 -2 0
// 8 12 16
// 9 13 17
//
// alpha * A          * op(B)   = alpha * C       =  C
// 0       6  10  14    5  4      0       14  4      0   0
// 1       7  11  15   -3 -2      1       17  6      17  6
// 2       8  12  16    1  0      2       20  8      40  16
// 3       9  13  17              3       23  10     69  30
//
// alpha * A          * op(B)   = alpha * C       =  C
// 1       6  10  14    5  4      1       14  4      14  4
//         7  11  15   -3 -2              17  6      17  6
//         8  12  16    1  0              20  8      20  8
//         9  13  17                      23  10     23  10
// clang-format on

int main() {
  bool pass = true;
  // test1 and test4 are moved to matmul_3.cu
  pass = test2() && pass;
  pass = test3() && pass;
  pass = test5() && pass;
  pass = test6() && pass;
  pass = test7() && pass;
  pass = test_version() && pass;

  if (pass)
    printf("matmul all passed.\n");

  return pass ? 0 : 1;
}
